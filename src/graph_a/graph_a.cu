#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <sstream>

#include "common/cuda_common.hpp"
#include "common/timelogger.hpp"

#define BENCHMARK 1

namespace YuriPerf {

TimeLogger g_logger;

// A = B * scale
__global__ void MatMul(float *A, float *B, int N, int scale) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if (i < N && j < N) {
    int idx = i * N + j;
    A[idx] = B[idx] * scale;
  }
}

// A = B + C
__global__ void MatAdd(float *A, float *B, float *C, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if (i < N && j < N) {
    int idx = i * N + j;
    A[idx] = B[idx] + C[idx];
  }
}


// This simulates the following graph execution:
// Input: Matrix A, scalars s1, ..., sN
// Multiply A by each scalar, saving the results in
// intermediate matrices B1, ..., BN
// Add all intermediate matrices together two by two,
// example: C1 = B1 + B2, C2 = C1 + B3, ..., C(N-1) = C(N-2) + BN
// in this case, the final result will be stored in CN
class GraphA {
public:
  GraphA(int N, int numScalars) : m_N(N), m_numScalars(numScalars) {
    // Allocate memory for the input matrix and scalars
    host_A = new float[m_N * m_N];
    scalars = new float[m_numScalars];

    m_A.newMem(m_N * m_N);

    // Allocate memory for the intermediate matrices (only on the device)
    m_B = new CudaMemory<float>[m_numScalars];
    for (int i = 0; i < m_numScalars; i++) {
      m_B[i].newMem(m_N * m_N);
    }

    // Allocate memory for the output matrix
    host_C = new float[m_N * m_N];
    m_C = new CudaMemory<float>[m_numScalars - 1];
    for (int i = 0; i < m_numScalars - 1; i++) {
      m_C[i].newMem(m_N * m_N);
    }

    // Initialize the input matrix and scalars
    for (int i = 0; i < m_N * m_N; i++) {
      host_A[i] = i;
    }
    for (int i = 0; i < m_numScalars; i++) {
      scalars[i] = i;
    }
  }
  ~GraphA() {
    delete[] host_A;
    delete[] host_C;
    delete[] scalars;
    delete[] m_B;
  }

  void exec(int iterations = 1) {
#if (BENCHMARK == 1)
    for (int i = 0; i < iterations; i++) {
      g_logger.startRecording("h2d");
#endif
      // Copy the input matrix and scalars to the GPU
      m_A.copyToDevice(host_A, m_N * m_N);
#if (BENCHMARK == 1)
      g_logger.stopRecording();
      // printf("Copy input to GPU: %ld us\n", duration(end - start));
#endif

      int numStreams = m_numScalars + 1;
      CudaStream *streams = new CudaStream[numStreams];

      // Dim3 sizes. Is this ok?
      dim3 dimBlock(32, 32);
      dim3 dimGrid((m_N + 31) / 32, (m_N + 31) / 32);

// #if (BENCHMARK == 1)
//       g_logger.startRecording("kernel");
// #endif

#if (BENCHMARK == 1)
      g_logger.startRecording("mul_kernel");
#endif
      // Multily the input matrix by each scalar
      for (int i = 0; i < m_numScalars; i++) {
        MatMul<<<dimGrid, dimBlock, 0, streams[i].get()>>>(
            m_B[i].get(), m_A.get(), m_N, scalars[i]);
      }

      hipDeviceSynchronize(); // Make sure all streams are done
#if (BENCHMARK == 1)
      g_logger.stopRecording();
      // printf("Multiply by scalars: %ld us\n", duration(end - start));
      g_logger.startRecording("add_kernel");
#endif

      // Add all intermediate matrices together two by two
      // First B1 + B2 = C1
      // Doing this synchronously, as we need the result of this operation
      MatAdd<<<dimGrid, dimBlock>>>(m_C[0].get(), m_B[0].get(), m_B[1].get(),
                                    m_N);

      // Then C1 + B3 = C2, and so on
      for (int i = 1; i < m_numScalars - 1; i++) {
        MatAdd<<<dimGrid, dimBlock>>>(m_C[i].get(), m_B[i + 1].get(),
                                      m_C[i - 1].get(), m_N);
      }

      hipDeviceSynchronize(); // Make sure all streams are done

// #if (BENCHMARK == 1)
//       g_logger.stopRecording();
// #endif

#if (BENCHMARK == 1)
      g_logger.stopRecording();
      // printf("Add intermediate matrices: %ld us\n", duration(end - start));
      g_logger.startRecording("d2h");
#endif

      // Copy the output matrix back to the host
      m_C[m_numScalars - 2].copyToHost(host_C, m_N * m_N);

#if (BENCHMARK == 1)
      g_logger.stopRecording();
      // printf("Copy output to CPU: %ld us\n", duration(end - start));
#endif
      delete[] streams;
    }
  }

  void print() {
    for (int i = 0; i < m_N; i++) {
      for (int j = 0; j < m_N; j++) {
        printf("%.2f\t", host_C[i * m_N + j]);
      }
      printf("\n");
    }
    printf("\n");
  }

private:
  int m_N;
  int m_numScalars;
  float *host_A;
  float *host_C;
  float *scalars;
  CudaMemory<float> m_A;
  CudaMemory<float> *m_B;
  CudaMemory<float> *m_C;
};

} // namespace YuriPerf

int main(int argc, char *argv[]) {
  int N = argc > 1 ? atoi(argv[1]) : 8;
  int numScalars = argc > 2 ? atoi(argv[2]) : 2;
  int iterations = argc > 3 ? atoi(argv[3]) : 1;

  std::stringstream fstr;
  fstr << "time_g1_" << numScalars << "_" << N << ".csv";

  std::string filename = argc > 4 ? argv[4] : fstr.str();

#if (BENCHMARK == 1)
  YuriPerf::g_logger.setActive(true);

  YuriPerf::g_logger.startProgram();
#endif

  // Create the graph
  YuriPerf::GraphA graph(N, numScalars);
  graph.exec(iterations);
  // graph.print();

#if (BENCHMARK == 1)
  YuriPerf::g_logger.endProgram();

  YuriPerf::g_logger.print();
  YuriPerf::g_logger.writeCSV(filename);
#endif

  return 0;
}
